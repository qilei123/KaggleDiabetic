#include "hip/hip_runtime.h"
//This does not really implement a convolution. It just gathers data together in prepartion for matrix muliplictation. "Proper convolution" = "ConvolutionalLayer" + "NetworkInNetworkLayer"

#include "ConvolutionalLayer.h"
#include <iostream>
#include <vector>
#include <cassert>
#include "cudaUtilities.h"
#include "utilities.h"
#include "Regions.h"

__global__ void dPropForwardToMatrixMultiplyInput
(float* d_features, float* d_convolved, int* rules, int count, int nIn) {
  __shared__ int r[KERNELBLOCKSIZE];
  for (int c=blockIdx.x*KERNELBLOCKSIZE; c<count; c+=(1<<12)*KERNELBLOCKSIZE) {
    int n=min(KERNELBLOCKSIZE,count-c);
    r[threadIdx.x]=(threadIdx.x<n)?rules[c+threadIdx.x]*nIn:0;
    __syncthreads();
    for (int q=0;q<n;q++) {
      int rq=r[q];
      int i=(c+q)*nIn;
      for (int j=threadIdx.x;j<nIn;j+=KERNELBLOCKSIZE) {
        d_convolved[i+j]=(rq>=0)?d_features[rq+j]:0;    //If padding is used, check rq!=-1
      }
    }
    __syncthreads();
  }
}
void propForwardToMatrixMultiply(float* inFeatures, float* outFeatures, int* rules, int count, int nIn) {
  assert(count>0);
  int batch=min(1<<12,(count+KERNELBLOCKSIZE-1)/KERNELBLOCKSIZE);
  dPropForwardToMatrixMultiplyInput<<<batch,KERNELBLOCKSIZE,0,cnnMemStream->stream>>>
    (inFeatures,
     outFeatures,
     rules,
     count, nIn);
  cudaCheckError();
}
__global__ void dPropBackwardFromMatrixMultiplyOutput
(float* d_deltaGrid, float* d_deltaConvolved, int* rules, int count, int nIn) {
  __shared__ int r[KERNELBLOCKSIZE];
  for (int c=blockIdx.x*KERNELBLOCKSIZE; c<count; c+=(1<<12)*KERNELBLOCKSIZE) {
    int n=min(KERNELBLOCKSIZE,count-c);
    r[threadIdx.x]=(threadIdx.x<n)?rules[c+threadIdx.x]*nIn:0;
    __syncthreads();
    for (int q=0;q<n;q++) {
      int rq=r[q];
      int i=(c+q)*nIn;
      for (int j=threadIdx.x;j<nIn;j+=KERNELBLOCKSIZE) {
        if (/*d_deltaConvolved[i+j]!=0 and*/ rq>=0)
          atomicAdd(&d_deltaGrid[rq+j],d_deltaConvolved[i+j]);
      }
    }
    __syncthreads();
  }
}
void propBackwardFromMatrixMultiply(float* inDFeatures, float* outDFeatures, int* rules, int count, int nIn) {
  assert(count>0);
  int batch=min(1<<12,(count+KERNELBLOCKSIZE-1)/KERNELBLOCKSIZE);
  dPropBackwardFromMatrixMultiplyOutput<<<batch,KERNELBLOCKSIZE,0,cnnMemStream->stream>>>
    (inDFeatures,
     outDFeatures,
     rules,
     count, nIn);
  cudaCheckError();
}

template <typename t> void convolutionFeaturesPresent(std::vector<t>& d_src, std::vector<t>& d_dest, int nf, int nfp, int nCopies) {
  for (int i=0;i<nfp*nCopies;++i) {
    d_dest[i]=d_src[i%nfp]+nf*(i/nfp);
  }
}
template void convolutionFeaturesPresent<int>(std::vector<int>& d_src, std::vector<int>& d_dest, int nf, int nfp, int nCopies);

ConvolutionalLayer::ConvolutionalLayer(int filterSize,
                                       int filterStride,
                                       int dimension,
                                       int nFeaturesIn,
                                       int minActiveInputs) :
  filterSize(filterSize),
  filterStride(filterStride),
  dimension(dimension),
  nFeaturesIn(nFeaturesIn),
  minActiveInputs(minActiveInputs) {
  fs=ipow(filterSize,dimension);
  nFeaturesOut=fs*nFeaturesIn;
  std::cout << "Convolution "
            << filterSize <<"^" <<dimension<< "x"<< nFeaturesIn
            << "->" << nFeaturesOut;
  if (filterStride>1)
    std::cout << " stride:" << filterStride;
  if (minActiveInputs>1)
    std::cout << " minActiveInputs:"  << minActiveInputs;
  std::cout << std::endl;
  }
void ConvolutionalLayer::preprocess
(SpatiallySparseBatch &batch,
 SpatiallySparseBatchInterface &input,
 SpatiallySparseBatchInterface &output) {
  output.nFeatures=nFeaturesOut;
  assert(input.nFeatures==nFeaturesIn);
  assert(input.spatialSize>=filterSize);
  assert((input.spatialSize-filterSize)%filterStride==0);
  output.spatialSize=(input.spatialSize-filterSize)/filterStride+1;
  output.nSpatialSites=0;
  output.grids.resize(batch.batchSize);
  output.backpropErrors=input.backpropErrors;
  RegularPoolingRegions regions(inSpatialSize, outSpatialSize,dimension,filterSize, filterStride);
  for (int item=0;item<batch.batchSize;item++) {
    gridRules(input.grids[item],
              output.grids[item],
              regions,
              output.nSpatialSites,
              output.rules.hVector(),
              minActiveInputs);
  }
  output.featuresPresent.copyToCPU();
  output.featuresPresent.resize(input.featuresPresent.size()*fs);
  convolutionFeaturesPresent(input.featuresPresent.hVector(), output.featuresPresent.hVector(), input.nFeatures, input.featuresPresent.size(), fs);
}
void ConvolutionalLayer::forwards
(SpatiallySparseBatch &batch,
 SpatiallySparseBatchInterface &input,
 SpatiallySparseBatchInterface &output) {
  output.sub->features.resize(output.nSpatialSites*output.featuresPresent.size());
  propForwardToMatrixMultiply(input.sub->features.dPtr(),
                              output.sub->features.dPtr(),
                              output.rules.dPtr(),
                              output.nSpatialSites*fs,
                              input.featuresPresent.size());
}
void ConvolutionalLayer::backwards
(SpatiallySparseBatch &batch,
 SpatiallySparseBatchInterface &input,
 SpatiallySparseBatchInterface &output,
 float learningRate,
 float momentum) {
  if (input.backpropErrors) {
    input.sub->dfeatures.resize(input.nSpatialSites*input.featuresPresent.size());
    input.sub->dfeatures.setZero(*cnnMemStream);
    propBackwardFromMatrixMultiply(input.sub->dfeatures.dPtr(),
                                   output.sub->dfeatures.dPtr(),
                                   output.rules.dPtr(),
                                   output.nSpatialSites*fs,
                                   input.featuresPresent.size());
    // output.sub->features.resize(0);
    // output.sub->dfeatures.resize(0);
    // cudaCheckError();
  }
}
int ConvolutionalLayer::calculateInputSpatialSize(int outputSpatialSize) {
  outSpatialSize=outputSpatialSize;
  inSpatialSize=filterSize+(outputSpatialSize-1)*filterStride;
  return inSpatialSize;
}
