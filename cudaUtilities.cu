#include "hip/hip_runtime.h"
#define CUDAUTILITIES_CU
#include "cudaUtilities.h"
#include <iostream>
#include <cassert>

static void cublasError(hipblasStatus_t error, const char* file, int linenumber)
{
  switch (error)
    {
    case HIPBLAS_STATUS_SUCCESS:
      break;

    case HIPBLAS_STATUS_NOT_INITIALIZED:
      std::cout << file << " " << linenumber<<std::endl;
      std::cout <<  "HIPBLAS_STATUS_NOT_INITIALIZED\n";
      break;

    case HIPBLAS_STATUS_ALLOC_FAILED:
      std::cout << file << " " << linenumber<<std::endl;
      std::cout <<  "HIPBLAS_STATUS_ALLOC_FAILED\n";
      break;

    case HIPBLAS_STATUS_INVALID_VALUE:
      std::cout << file << " " << linenumber<<std::endl;
      std::cout <<  "HIPBLAS_STATUS_INVALID_VALUE\n";
      break;

    case HIPBLAS_STATUS_ARCH_MISMATCH:
      std::cout << file << " " << linenumber<<std::endl;
      std::cout <<  "HIPBLAS_STATUS_ARCH_MISMATCH\n";
      break;

    case HIPBLAS_STATUS_MAPPING_ERROR:
      std::cout << file << " " << linenumber<<std::endl;
      std::cout <<  "HIPBLAS_STATUS_MAPPING_ERROR\n";
      break;

    case HIPBLAS_STATUS_EXECUTION_FAILED:
      std::cout << file << " " << linenumber<<std::endl;
      std::cout <<  "HIPBLAS_STATUS_EXECUTION_FAILED\n";
      break;

    case HIPBLAS_STATUS_INTERNAL_ERROR:
      std::cout << file << " " << linenumber<<std::endl;
      std::cout <<  "HIPBLAS_STATUS_INTERNAL_ERROR\n";
      break;
    }
}

int intRoundUp(int a, int d) {
  return ((a+d-1)/d)*d;
}
int intRound(int a, int d) {
  return round(a*1.0/d)*d;
}

int initializeGPU(int pciBusID) { //pciBusID, or -1 for the first device
  int nGPU;
  int deviceID=-1;
  cudaSafeCall(hipGetDeviceCount(&nGPU));
  for (int i=0;i<nGPU;i++) {
    hipDeviceProp_t prop;
    cudaSafeCall(hipGetDeviceProperties(&prop, i));
    if (i==0 and pciBusID==-1)
      pciBusID=prop.pciBusID;
    if (prop.pciBusID==pciBusID) {
      std::cout << "*";
      cudaSafeCall(hipSetDevice(i));
      deviceID=i;
    } else {
      std::cout << " ";
    }
    std::cout << prop.pciBusID << " " << prop.name<< " " << (prop.totalGlobalMem>>20) << "MB Compute capability: " << prop.major << "." << prop.minor << std::endl;
  }
  assert(deviceID>=0);
  cublasError(hipblasCreate(&cublasHandle),__FILE__,__LINE__);
  cnnMemStream = new cudaMemStream();
  cublasError(hipblasSetStream(cublasHandle, cnnMemStream->stream));
  return deviceID;
}
//////////////////////////////////////////////////////////////////////////////////////////////////
//GEMM for matrices in row major form. ///////////////////////////////////////////////////////////
//A is l*m, B is m*r, C is l*r. Set C to alpha A B + beta C.
void d_rowMajorSGEMM_alphaAB_betaC (hipblasHandle_t handle,
                                    float* A, float* B, float* C,
                                    int l, int m, int r,
                                    float alpha, float beta, const char* file, int linenumber)
{
  cublasError(hipblasSgemm (handle, HIPBLAS_OP_N, HIPBLAS_OP_N,r,l,m,&alpha,B,r,A,m,&beta,C,r), file, linenumber);
}
//A^t is l*m, B is m*r, C is l*r
void d_rowMajorSGEMM_alphaAtB_betaC (hipblasHandle_t handle,
                                     float* A, float* B, float* C,
                                     int l, int m, int r,
                                     float alpha, float beta, const char* file, int linenumber)
{
  cublasError(hipblasSgemm (handle, HIPBLAS_OP_N, HIPBLAS_OP_T,r,l,m,&alpha,B,r,A,l,&beta,C,r), file, linenumber);
}
//A is l*m, B^t is m*r, C is l*r
void d_rowMajorSGEMM_alphaABt_betaC (hipblasHandle_t handle,
                                     float* A, float* B, float* C,
                                     int l, int m, int r,
                                     float alpha, float beta, const char* file, int linenumber)
{
  cublasError(hipblasSgemm (handle, HIPBLAS_OP_T, HIPBLAS_OP_N,r,l,m,&alpha,B,m,A,m,&beta,C,r), file, linenumber);
}
//A^t is l*m, B^t is m*r, C is l*r
void d_rowMajorSGEMM_alphaAtBt_betaC (hipblasHandle_t handle,
                                      float* A, float* B, float* C,
                                      int l, int m, int r,
                                      float alpha, float beta, const char* file, int linenumber)
{
  cublasError(hipblasSgemm (handle, HIPBLAS_OP_T, HIPBLAS_OP_T,r,l,m,&alpha,B,m,A,l,&beta,C,r), file, linenumber);
}


cudaMemStream::cudaMemStream() : pinnedMemorySize(1<<24) {
  cudaSafeCall(hipHostMalloc(&pinnedMemory,pinnedMemorySize));
  cudaSafeCall(hipStreamCreate(&stream));
}
cudaMemStream::~cudaMemStream() {
  cudaSafeCall(hipStreamDestroy(stream));
  hipHostFree(pinnedMemory);
}
